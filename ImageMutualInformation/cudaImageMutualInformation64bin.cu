#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// CUDA includes
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include "mutual_information_common.h"


#define LOG_2 std::log(2.0)

////////////////////////////////////////////////////////////////////////////////
// Shortcut shared memory atomic addition functions
////////////////////////////////////////////////////////////////////////////////
inline __device__ void addByte(uint *s_JointHist, uint *s_WarpHist1, uint *s_WarpHist2, uint data1, uint data2)
{
  uint d1 = data1 / 4;
  uint d2 = data2 / 4;
  atomicAdd(s_WarpHist1 + d1, 1);
  atomicAdd(s_WarpHist2 + d2, 1);
  atomicAdd(s_JointHist + d1 * HISTOGRAM64_BIN_COUNT + d2, 1);
}

inline __device__ void addWord(uint *s_JointHist, uint *s_WarpHist1, uint *s_WarpHist2, uint data1, uint data2)
{
  addByte(s_JointHist, s_WarpHist1, s_WarpHist2, (data1 >> 0) & 0xFFU, (data2 >>  0) & 0xFFU);
  addByte(s_JointHist, s_WarpHist1, s_WarpHist2, (data1 >> 8) & 0xFFU, (data2 >>  8) & 0xFFU);
  addByte(s_JointHist, s_WarpHist1, s_WarpHist2, (data1 >> 16) & 0xFFU, (data2 >>  16) & 0xFFU);
  addByte(s_JointHist, s_WarpHist1, s_WarpHist2, (data1 >> 24) & 0xFFU, (data2 >>  24) & 0xFFU);
}


__global__ void histogram64Kernel(uint *d_PartialJointHistograms,
				  uint *d_PartialHistograms1, uint *d_PartialHistograms2,
				  uint *d_Data1, uint *d_Data2, uint dataCount)
{
  __shared__ uint s_Hist1[HISTOGRAM64_THREADBLOCK_MEMORY]; //  warp_count * 64
  __shared__ uint s_Hist2[HISTOGRAM64_THREADBLOCK_MEMORY]; //  warp_count * 64
  __shared__ uint s_JointHist[JOINT_HISTOGRAM64_THREADBLOCK_MEMORY]; // 64 * 64
 
  // calculating starting position for the warp
  uint *s_WarpHist1 = s_Hist1 + (threadIdx.x >> LOG2_WARP_SIZE) * HISTOGRAM64_BIN_COUNT;
  uint *s_WarpHist2 = s_Hist2 + (threadIdx.x >> LOG2_WARP_SIZE) * HISTOGRAM64_BIN_COUNT;

  
  // Clear shared memory storage for current threadblock before processing
#pragma unroll
  // memory = 6 * 64  ; block size = 6 * 32
  // i < 2
  for (uint i = 0; i < (HISTOGRAM64_THREADBLOCK_MEMORY / HISTOGRAM64_THREADBLOCK_SIZE); i++)
    {
      s_Hist1[threadIdx.x + i * HISTOGRAM64_THREADBLOCK_SIZE] = 0;
      s_Hist2[threadIdx.x + i * HISTOGRAM64_THREADBLOCK_SIZE] = 0;
    }

  for (uint i = 0; i < (JOINT_HISTOGRAM64_THREADBLOCK_MEMORY / HISTOGRAM64_THREADBLOCK_SIZE); i++)
    {
      s_JointHist[threadIdx.x + i * HISTOGRAM64_THREADBLOCK_SIZE] = 0;
    }
  __syncthreads();

  for (uint pos = UMAD(blockIdx.x, blockDim.x, threadIdx.x); pos < dataCount; pos += UMUL(blockDim.x, gridDim.x))
    {
      uint data1 = d_Data1[pos];
      uint data2 = d_Data2[pos];
      addWord(s_JointHist, s_WarpHist1, s_WarpHist2, data1, data2);
    }
  // Merge per-warp histograms into per-block and write to global memory
  __syncthreads();

  for (uint bin = threadIdx.x; bin < HISTOGRAM64_BIN_COUNT; bin += HISTOGRAM64_THREADBLOCK_SIZE)
    {
      uint sum1 = 0;
      uint sum2 = 0;

      for (uint i = 0; i < WARP_COUNT; i++)
	{
	  sum1 += s_Hist1[bin + i * HISTOGRAM64_BIN_COUNT];
	  sum2 += s_Hist2[bin + i * HISTOGRAM64_BIN_COUNT];
	}
      // per block sub-histogram 
      d_PartialHistograms1[blockIdx.x * HISTOGRAM64_BIN_COUNT + bin] = sum1;
      d_PartialHistograms2[blockIdx.x * HISTOGRAM64_BIN_COUNT + bin] = sum2;  
    }
  
  // store shared Joint histogram on global mem
  for (uint i = 0; i < (JOINT_HISTOGRAM64_THREADBLOCK_MEMORY / HISTOGRAM64_THREADBLOCK_SIZE); i++)
    {
      d_PartialJointHistograms[blockIdx.x * JOINT_HISTOGRAM64_BIN_COUNT + threadIdx.x + i * HISTOGRAM64_THREADBLOCK_SIZE]
	= s_JointHist[threadIdx.x + i * HISTOGRAM64_THREADBLOCK_SIZE];
    }
}
////////////////////////////////////////////////////////////////////////////////
// Merge histogram64() output
// Run one threadblock per bin; each threadblock adds up the same bin counter
// from every partial histogram. Reads are uncoalesced, but mergeHistogram256
// takes only a fraction of total processing time
////////////////////////////////////////////////////////////////////////////////
#define MERGE_THREADBLOCK_SIZE 1024

__global__ void mergeHistogram64Kernel(
				       uint *d_Histogram1,
				       uint *d_Histogram2,
				       uint *d_PartialHistograms1,
				       uint *d_PartialHistograms2,
				       uint histogramCount )
{
  uint sum1 = 0;
  uint sum2 = 0;
  
  for (uint i = threadIdx.x; i < histogramCount; i += MERGE_THREADBLOCK_SIZE)
    {
      sum1 += d_PartialHistograms1[blockIdx.x + i * HISTOGRAM64_BIN_COUNT];
      sum2 += d_PartialHistograms2[blockIdx.x + i * HISTOGRAM64_BIN_COUNT];
    }

  __shared__ uint data1[MERGE_THREADBLOCK_SIZE];
  __shared__ uint data2[MERGE_THREADBLOCK_SIZE];
  
  data1[threadIdx.x] = sum1;
  data2[threadIdx.x] = sum2;

  for (uint stride = MERGE_THREADBLOCK_SIZE / 2; stride > 0; stride >>= 1)
    {
      __syncthreads();
      // reduction
      if (threadIdx.x < stride)
        {
	  data1[threadIdx.x] += data1[threadIdx.x + stride];
	  data2[threadIdx.x] += data2[threadIdx.x + stride];
        }
    }
  // blockIdx is the bin number.
  if (threadIdx.x == 0)
    {
      d_Histogram1[blockIdx.x] = data1[0];
      d_Histogram2[blockIdx.x] = data2[0];
    }
}


__global__ void mergeJointHistogram64Kernel(uint *d_JointHistogram, uint *d_PartialHistograms, uint jointHistogramCount )
{
  double sum = 0;
  for (uint i = threadIdx.x; i < jointHistogramCount; i += MERGE_THREADBLOCK_SIZE)
    {
      sum += d_PartialHistograms[blockIdx.x + i * JOINT_HISTOGRAM64_BIN_COUNT];
    }

  __shared__ uint data[MERGE_THREADBLOCK_SIZE];

  data[threadIdx.x] = sum;

  for (uint stride = MERGE_THREADBLOCK_SIZE / 2; stride > 0; stride >>= 1)
    {
      __syncthreads();
      if (threadIdx.x < stride)
	{
	  data[threadIdx.x] += data[threadIdx.x + stride];
	}
    }
  if (threadIdx.x == 0)
    {
      d_JointHistogram[blockIdx.x] = data[0];
    }
}


static const uint PARTIAL_HISTOGRAM64_COUNT = 240;
static uint *d_PartialHistograms1;
static uint *d_PartialHistograms2;
static uint *d_PartialJointHistograms;
static double *d_PartialJointEntropy;
static double *d_ImageEntropy1;
static double *d_ImageEntropy2;
static double *d_JointEntropy;


//Internal memory allocation
extern "C" void initHistogram64(void)
{
  //d_PartialHistogram is a uint array of size 240 * 64,
  checkCudaErrors(hipMalloc((void **)&d_PartialHistograms1, PARTIAL_HISTOGRAM64_COUNT * HISTOGRAM64_BIN_COUNT * sizeof(uint)));
  checkCudaErrors(hipMalloc((void **)&d_PartialHistograms2, PARTIAL_HISTOGRAM64_COUNT * HISTOGRAM64_BIN_COUNT * sizeof(uint)));
  checkCudaErrors(hipMalloc((void **)&d_PartialJointHistograms, PARTIAL_HISTOGRAM64_COUNT * JOINT_HISTOGRAM64_BIN_COUNT * sizeof(uint)));
  checkCudaErrors(hipMalloc((void **)&d_PartialJointEntropy, HISTOGRAM64_BIN_COUNT * sizeof(double)));
  checkCudaErrors(hipMalloc((void **)&d_ImageEntropy1, sizeof(double)));
  checkCudaErrors(hipMalloc((void **)&d_ImageEntropy2, sizeof(double)));
  checkCudaErrors(hipMalloc((void **)&d_JointEntropy, sizeof(double)));
 
}

//Internal memory deallocation
extern "C" void closeHistogram64(void)
{
  checkCudaErrors(hipFree(d_PartialHistograms1));
  checkCudaErrors(hipFree(d_PartialHistograms2));
  checkCudaErrors(hipFree(d_PartialJointHistograms));
  checkCudaErrors(hipFree(d_PartialJointEntropy));
}


extern "C" void histogram64(
			    uint *d_JointHistogram,
			    uint *d_Histogram1,
			    uint *d_Histogram2,
			    void *d_Data1,
			    void *d_Data2,
			    uint byteCount1,
			    uint byteCount2)
{
  // assert(byteCount % sizeof(uint) == 0);

  uint byteCount = (byteCount1 < byteCount2) ? byteCount1 : byteCount2;
  
  histogram64Kernel<<<PARTIAL_HISTOGRAM64_COUNT, HISTOGRAM64_THREADBLOCK_SIZE>>>(   d_PartialJointHistograms,
										    d_PartialHistograms1,
										    d_PartialHistograms2,
										    (uint *)d_Data1,
										    (uint *)d_Data2,
										    byteCount
										    );
  getLastCudaError("histogram64Kernel() execution failed\n");
  

  mergeHistogram64Kernel<<<HISTOGRAM64_BIN_COUNT, MERGE_THREADBLOCK_SIZE>>>(
									    d_Histogram1,
									    d_Histogram2,
									    d_PartialHistograms1,
									    d_PartialHistograms2,
									    PARTIAL_HISTOGRAM64_COUNT
									    );
  getLastCudaError("mergeHistogram64Kernel() execution failed\n");


  mergeJointHistogram64Kernel<<<JOINT_HISTOGRAM64_BIN_COUNT, MERGE_THREADBLOCK_SIZE>>>(
										       d_JointHistogram,
										       d_PartialJointHistograms,
										       PARTIAL_HISTOGRAM64_COUNT
										       );
  getLastCudaError("mergeJointHistogram64Kernel() execution failed\n");
  
}


__global__ void entropy_kernel(double *d_ImageEntropy1,
			       double *d_ImageEntropy2,
			       uint *d_Histogram1,
			       uint *d_Histogram2,
			       uint totalCount)
{
  // calculate entropy 1 in block 0
  if (blockIdx.x == 0)
    {
      __shared__ double s_entropy1[HISTOGRAM64_BIN_COUNT];
      uint tid = threadIdx.x;
  
      if(d_Histogram1[tid])
	s_entropy1[tid] = - ((double)d_Histogram1[tid] / totalCount) * std::log((double)d_Histogram1[tid] / totalCount ) / LOG_2;
      else
	s_entropy1[tid] = 0;
      // before reduce, make sure threads within block finish its own work.
      __syncthreads();
      // reduction method from CUDA sample reduction code ( most optimized )
      if (tid < 32) s_entropy1[tid] += s_entropy1[tid + 32];	  
      if (tid < 16) s_entropy1[tid] += s_entropy1[tid + 16];	  
      if (tid < 8) s_entropy1[tid] += s_entropy1[tid + 8];	  
      if (tid < 4) s_entropy1[tid] += s_entropy1[tid + 4];	  
      if (tid < 2) s_entropy1[tid] += s_entropy1[tid + 2];	  
      if (tid < 1) s_entropy1[tid] += s_entropy1[tid + 1];	 
      if(tid == 0) d_ImageEntropy1[0] = s_entropy1[0];
    }
  // calculate entropy 2 in block 1
  if (blockIdx.x == 1)
    {
      __shared__ double s_entropy2[HISTOGRAM64_BIN_COUNT];
      uint tid = threadIdx.x;
      
      if(d_Histogram2[tid])
	s_entropy2[tid] = - ((double)d_Histogram2[tid] / totalCount) * std::log((double)d_Histogram2[tid] / totalCount) / LOG_2;
      else
	s_entropy2[tid] = 0;
      // before reduce, make sure threads within block finish its own work.
      __syncthreads();
      // reduction 
      if (tid < 32) s_entropy2[tid] += s_entropy2[tid + 32];
      if (tid < 16) s_entropy2[tid] += s_entropy2[tid + 16];
      if (tid < 8) s_entropy2[tid] += s_entropy2[tid + 8];
      if (tid < 4) s_entropy2[tid] += s_entropy2[tid + 4];
      if (tid < 2) s_entropy2[tid] += s_entropy2[tid + 2];
      if (tid < 1) s_entropy2[tid] += s_entropy2[tid + 1];
      if(tid == 0) d_ImageEntropy2[0] = s_entropy2[0];
    }
}


__global__ void joint_entropy_kernel(double *d_PartialJointEntropy, uint *d_JointHistogram, uint totalCount)
{
  __shared__ double s_joint_entropy[HISTOGRAM64_BIN_COUNT];
  uint tid = threadIdx.x;
  uint bid = blockIdx.x;
  
  uint i = bid * blockDim.x + threadIdx.x;
      
  s_joint_entropy[tid] = (d_JointHistogram[i] == 0) ?
    0 :  - ((double)d_JointHistogram[i] / totalCount) * std::log((double)d_JointHistogram[i] / totalCount) / LOG_2;
 
  __syncthreads();
  // reduce
  if (tid < 32) s_joint_entropy[tid] += s_joint_entropy[tid + 32];
  if (tid < 16) s_joint_entropy[tid] += s_joint_entropy[tid + 16];
  if (tid < 8) s_joint_entropy[tid] += s_joint_entropy[tid + 8];
  if (tid < 4) s_joint_entropy[tid] += s_joint_entropy[tid + 4];
  if (tid < 2) s_joint_entropy[tid] += s_joint_entropy[tid + 2];
  if (tid < 1) s_joint_entropy[tid] += s_joint_entropy[tid + 1];
  if(tid == 0) d_PartialJointEntropy[bid] = s_joint_entropy[0];
 
}


// global memory is coalesced 
__global__ void merge_joint_entropy_kernel(double *d_JointEntropy, double *d_PartialJointEntropy)
{
  uint tid = threadIdx.x;
  if (tid < 32) 
    d_PartialJointEntropy[tid] += d_PartialJointEntropy[tid + 32];
  if (tid < 16)
    d_PartialJointEntropy[tid] += d_PartialJointEntropy[tid + 16];
  if (tid < 8)
    d_PartialJointEntropy[tid] += d_PartialJointEntropy[tid + 8];
  if (tid < 4)
    d_PartialJointEntropy[tid] += d_PartialJointEntropy[tid + 4];
  if (tid < 2)
    d_PartialJointEntropy[tid] += d_PartialJointEntropy[tid + 2];
  if (tid < 1)
    d_PartialJointEntropy[tid] += d_PartialJointEntropy[tid + 1];
  if (tid == 0)
    d_JointEntropy[0] = d_PartialJointEntropy[0];
}


extern "C" void getImageEntropyAndJointEntropy(uint *d_Histogram1,
					       uint *d_Histogram2,
					       uint *d_JointHistogram,
					       uint commonPixelCount)
{
  // puts("getImageEntropyAndJointEntropy...");
   
  // puts("entering entropy_kernel...");
  entropy_kernel<<<2, 64>>>(d_ImageEntropy1, d_ImageEntropy2, d_Histogram1, d_Histogram2, commonPixelCount);

  
   
  //  puts("entering joint_entropy_kernel...");
  joint_entropy_kernel<<<64, 64>>>(d_PartialJointEntropy, d_JointHistogram, commonPixelCount);


  // puts("merging joint_entropy kernel...");
  merge_joint_entropy_kernel<<<1 , 64 >>>(d_JointEntropy,  d_PartialJointEntropy);

}



extern "C" bool cudaImageMutualInformation64( double *h_JointEntropy,
					      double *h_Entropy1,
					      double *h_Entropy2,
					      uint *h_JointHistogram,
					      uint *h_Histogram1,
					      uint *h_Histogram2,
					      uchar *h_Data1,
					      uint dataCount1,
					      uchar *h_Data2,
					      uint dataCount2)
{
  hipEvent_t start_device, stop_device, start_histogram, stop_histogram, start_entropy, stop_entropy;
  
  hipEventCreate(&start_device);
  hipEventCreate(&stop_device);
  hipEventCreate(&start_histogram);
  hipEventCreate(&stop_histogram);
  hipEventCreate(&start_entropy);
  hipEventCreate(&stop_entropy);
  
  hipEventRecord(start_device,0);
  
  uchar *d_Data1, *d_Data2;
  uint *d_JointHistogram, *d_Histogram1, *d_Histogram2;
  
  uint byteCount1, byteCount2;
  //  uint copyCount1, copyCount2;

  //  uint countRemainder1 = dataCount1 % sizeof(uint);
  //  uint countRemainder2 = dataCount2 % sizeof(uint);
  
  byteCount1 = dataCount1 / sizeof(uint);
  byteCount2 = dataCount2 / sizeof(uint);
  // copyCount1 = dataCount1 - countRemainder1;
  // copyCount2 = dataCount2 - countRemainder2;
  
  //  printf(">>>>\t Allocating GPU Memory\t <<<<\n");
  checkCudaErrors(hipMalloc((void **)&d_Data1, dataCount1));
  checkCudaErrors(hipMalloc((void **)&d_Data2, dataCount2));
  
  checkCudaErrors(hipMalloc((void **)&d_Histogram1,			
			     HISTOGRAM64_BIN_COUNT * sizeof(uint)));
  
  checkCudaErrors(hipMalloc((void **)&d_Histogram2,
			     HISTOGRAM64_BIN_COUNT * sizeof(uint)));

  checkCudaErrors(hipMalloc((void **)&d_JointHistogram,
			     HISTOGRAM64_BIN_COUNT * HISTOGRAM64_BIN_COUNT * sizeof(uint)));
  
  
  // printf(">>>>\t Copying Input Data\t <<<<\n");
  checkCudaErrors(hipMemcpy(d_Data1, h_Data1, dataCount1, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_Data2, h_Data2, dataCount2, hipMemcpyHostToDevice));

  
  // printf(">>>>\t Initializing 64-bin histogram\t <<<<\n");
  initHistogram64();

  hipEventRecord(start_histogram, 0);
  
  // histogram calculation
  histogram64(d_JointHistogram, d_Histogram1, d_Histogram2, d_Data1, d_Data2, byteCount1, byteCount2);
  
  hipDeviceSynchronize();
  
  hipEventRecord(stop_histogram,0);
  hipEventSynchronize(stop_histogram);
  
  float histogram_time;
  hipEventElapsedTime(&histogram_time, start_histogram, stop_histogram);
  
  ////////////////////////////////////////////////////////////////////////////////////////////////////////////
  // now we have all 3 histograms, i.e. histogram of image A, histogram of image B, joint histogram of A and B.
  // calculate image entropy of each.
  ////////////////////////////////////////////////////////////////////////////////////////////////////////////

  uint commonPixelCount = (dataCount1 < dataCount2) ? dataCount1 : dataCount2;

  hipEventRecord(start_entropy, 0);
  // calculate entropy and joint entropy
  getImageEntropyAndJointEntropy( d_Histogram1, d_Histogram2, d_JointHistogram, commonPixelCount);
  
  hipEventRecord(stop_entropy,0);
  hipEventSynchronize(stop_entropy);
  
  float entropy_time;
  hipEventElapsedTime(&entropy_time, start_entropy, stop_entropy);
  
  
  checkCudaErrors(hipMemcpy(h_Entropy1, d_ImageEntropy1, sizeof(double), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_Entropy2, d_ImageEntropy2, sizeof(double), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_JointEntropy, d_JointEntropy, sizeof(double), hipMemcpyDeviceToHost));

  //printf(">>>>\t Returning GPU results \t<<<<\n");
  checkCudaErrors(hipMemcpy(h_Histogram1,
			     d_Histogram1,
			     HISTOGRAM64_BIN_COUNT * sizeof(uint),	
			     hipMemcpyDeviceToHost));

  checkCudaErrors(hipMemcpy(h_Histogram2,
			     d_Histogram2,
			     HISTOGRAM64_BIN_COUNT * sizeof(uint),	
			     hipMemcpyDeviceToHost));

  
  checkCudaErrors(hipMemcpy(h_JointHistogram,
			     d_JointHistogram,
			     HISTOGRAM64_BIN_COUNT * HISTOGRAM64_BIN_COUNT * sizeof(uint),	
			     hipMemcpyDeviceToHost));
 
  /*
    printf("countRemainder1 : %d \ncountRemainder2 : %d\n", countRemainder1, countRemainder2);  
    for (uint i = 0; i < countRemainder1; i++)
    h_Histogram1[ (uint) *(h_Data1 + copyCount1 + i) ]++;

    for (uint i = 0; i < countRemainder2; i++)
    h_Histogram2[ (uint) *(h_Data2 + copyCount2 + i) ]++;
    printf(">>>>\t Shutting down 64-bin histogram \t<<<<\n");
  */
  
  closeHistogram64();
  
  checkCudaErrors(hipFree(d_Data1));
  checkCudaErrors(hipFree(d_Histogram1));
  checkCudaErrors(hipFree(d_Data2));
  checkCudaErrors(hipFree(d_Histogram2));
  checkCudaErrors(hipFree(d_JointHistogram));

  hipEventRecord(stop_device,0);
  hipEventSynchronize(stop_device);
  float device_time;
  hipEventElapsedTime(&device_time, start_device, stop_device);

  printf("Histogram Calculation Time             =  %f ms\n\n", histogram_time );
  printf("Entropy Calculation Time               =  %f ms\n\n", entropy_time );
  printf("CUDA Device Total Running Time         =  %f ms\n\n", device_time );
  return 0;
}